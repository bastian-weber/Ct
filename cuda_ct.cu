#include "hip/hip_runtime.h"
#include "cuda_ct.h"

namespace ct {

	namespace cuda {

		int getMultiprocessorCnt(int deviceId) {
			int cnt;
			hipDeviceGetAttribute(&cnt, hipDeviceAttributeMultiprocessorCount, deviceId);
			return cnt;
		}
		hipPitchedPtr create3dVolumeOnGPU(size_t xSize, size_t ySize, size_t zSize, bool& success) {
			success = true;
			hipError_t status;
			hipExtent extent = make_hipExtent(xSize * sizeof(float), ySize, zSize);
			hipPitchedPtr ptr;
			status = hipMalloc3D(&ptr, extent);
			if (status != hipSuccess) {
				std::cout << "hipMalloc3D ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
			status = hipMemset3D(ptr, 0, extent);
			if (status != hipSuccess) {
				std::cout << "hipMemset3D ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
			return ptr;
		}

		void delete3dVolumeOnGPU(hipPitchedPtr devicePtr, bool& success) {
			success = true;
			hipError_t status = hipFree(devicePtr.ptr);
			if (status != hipSuccess) {
				std::cout << "hipFree ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
		}

		std::shared_ptr<float> download3dVolume(hipPitchedPtr devicePtr, size_t xSize, size_t ySize, size_t zSize, bool& success) {
			success = true;
			float* hostDataPtr = new float[xSize * ySize * zSize];
			hipPitchedPtr hostPtr = make_hipPitchedPtr(hostDataPtr, xSize * sizeof(float), xSize, ySize);
			hipExtent extent = make_hipExtent(xSize * sizeof(float), ySize, zSize);
			hipMemcpy3DParms memcopyParameters = { 0 };
			memcopyParameters.srcPtr = devicePtr;
			memcopyParameters.dstPtr = hostPtr;
			memcopyParameters.extent = extent;
			memcopyParameters.kind = hipMemcpyDeviceToHost;
			hipError_t status = hipMemcpy3D(&memcopyParameters);
			if (status != hipSuccess) {
				std::cout << "hipMemcpy3D ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
			return std::shared_ptr<float>(hostDataPtr, std::default_delete<float[]>());
		}

		__device__ float bilinearInterpolation(double u, double v, float u0v0, float u1v0, float u0v1, float u1v1) {
			//the two interpolations on the u axis
			double v0 = (1.0 - u)*u0v0 + u*u1v0;
			double v1 = (1.0 - u)*u0v1 + u*u1v1;
			//interpolation on the v axis between the two u-interpolated values
			return (1.0 - v)*v0 + v*v1;
		}

		__device__ void addToVolumeElement(hipPitchedPtr volumePtr, size_t ySize, size_t xCoord, size_t yCoord, size_t zCoord, float value) {
			char* devicePtr = (char*)(volumePtr.ptr);
			//z * xSize * ySize + y * xSize + x
			size_t pitch = volumePtr.pitch;
			size_t slicePitch = pitch * ySize;
			char* slice = devicePtr + zCoord*slicePitch;
			float* row = (float*)(slice + yCoord * pitch);
			row[xCoord] += value;
		}

		__global__ void reconstructionKernel(cv::cuda::PtrStepSz<float> image, cv::cuda::PtrStepSz<float> lookupTable, hipPitchedPtr volumePtr, size_t xSize, size_t ySize, size_t zSize, size_t zOffset, double radiusSquared, double sine, double cosine, double heightOffset, double uOffset, double SD, double imageLowerBoundU, double imageUpperBoundU, double imageLowerBoundV, double imageUpperBoundV, double imageToMatUPrecomputed, double imageToMatVPrecomputed) {

			size_t xIndex = threadIdx.x + blockIdx.x * blockDim.x;
			size_t yIndex = threadIdx.y + blockIdx.y * blockDim.y;
			size_t zIndex = threadIdx.z + blockIdx.z * blockDim.z;

			//if (xIndex == 0 && yIndex == 0 && zIndex == 0) {
			//	printf("kernel start\n");
			//}

			//just make sure we're inside the volume bounds
			if (xIndex < xSize && yIndex < ySize && zIndex < zSize) {

				//calculate the world coordinates
				double x = lookupTable(0, xIndex);
				double y = lookupTable(1, yIndex);
				double z = lookupTable(2, zIndex + zOffset);

				//check if voxel is inside the reconstructable cylinder
				if ((x*x + y*y) < radiusSquared) {

					double t = (-1)*x*sine + y*cosine;
					t += uOffset;
					double s = x*cosine + y*sine;
					double u = (t*SD) / (SD - s);
					double v = ((z + heightOffset)*SD) / (SD - s);

					//check if it's inside the image (before the coordinate transformation)
					if (u >= imageLowerBoundU && u <= imageUpperBoundU && v >= imageLowerBoundV && v <= imageUpperBoundV) {

						u += imageToMatUPrecomputed;
						v = (-1)*v + imageToMatVPrecomputed;

						//get the 4 surrounding pixels for bilinear interpolation (note: u and v are always positive)
						size_t u0 = u;
						size_t u1 = u0 + 1;
						size_t v0 = v;
						size_t v1 = v0 + 1;

						float u0v0 = image(v0, u0);
						float u1v0 = image(v0, u1);
						float u0v1 = image(v1, u0);
						float u1v1 = image(v1, u1);

						float value = bilinearInterpolation(u - double(u0), v - double(v0), u0v0, u1v0, u0v1, u1v1);

						addToVolumeElement(volumePtr, ySize, xIndex, yIndex, zIndex, value);
					}
				}
			}

			//if (xIndex == 0 && yIndex == 0 && zIndex == 0) {
			//	printf("kernel end\n");
			//}

		}

		void startReconstruction(cv::cuda::PtrStepSz<float> image, 
								 cv::cuda::PtrStepSz<float> lookupTable,
								 hipPitchedPtr volumePtr, 
								 size_t xSize, 
								 size_t ySize, 
								 size_t zSize, 
								 size_t zOffset, 
								 double radiusSquared, 
								 double sine, 
								 double cosine, 
								 double heightOffset, 
								 double uOffset, 
								 double SD, 
								 double imageLowerBoundU, 
								 double imageUpperBoundU, 
								 double imageLowerBoundV, 
								 double imageUpperBoundV, 
								 double imageToMatUPrecomputed, 
								 double imageToMatVPrecomputed, 
								 bool& success) {
			success = true;
			dim3 threads(32, 32, 1);
			dim3 blocks(((unsigned int)xSize + threads.x - 1) / threads.x,
						((unsigned int)ySize + threads.y - 1) / threads.y,
						((unsigned int)zSize + threads.z - 1) / threads.z);
			reconstructionKernel << < blocks, threads >> >(image,
														   lookupTable,
														   volumePtr,
														   xSize,
														   ySize,
														   zSize,
														   zOffset,
														   radiusSquared,
														   sine,
														   cosine,
														   heightOffset,
														   uOffset,
														   SD,
														   imageLowerBoundU,
														   imageUpperBoundU,
														   imageLowerBoundV,
														   imageUpperBoundV,
														   imageToMatUPrecomputed,
														   imageToMatVPrecomputed);
			hipError_t status = hipGetLastError();
			if (status != hipSuccess) {
				std::cout << std::endl << "Kernel launch ERROR: " << hipGetErrorString(status);
				success = false;
			}
		}

		void deviceSynchronize(bool& success) {
			success = true;
			hipError_t status = hipDeviceSynchronize();
			if (status != hipSuccess) {
				std::cout << std::endl << "hipDeviceSynchronize ERROR: " << hipGetErrorString(status);
				success = false;
			}
		}

	}

}