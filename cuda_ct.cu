#include "hip/hip_runtime.h"
#include "cuda_ct.h"

namespace ct {

	namespace cuda {

		hipPitchedPtr create3dVolumeOnGPU(size_t xSize, size_t ySize, size_t zSize) {
			hipExtent extent = make_hipExtent(xSize * sizeof(float), ySize, zSize);
			hipPitchedPtr ptr;
			hipMalloc3D(&ptr, extent);
			hipMemset3D(ptr, 0, extent);
			return ptr;
		}

		void delete3dVolumeOnGPU(hipPitchedPtr devicePtr) {
			hipFree(devicePtr.ptr);
		}

		std::shared_ptr<float> download3dVolume(hipPitchedPtr devicePtr, size_t xSize, size_t ySize, size_t zSize) {
			float* hostDataPtr = new float[xSize * ySize * zSize];
			hipPitchedPtr hostPtr = make_hipPitchedPtr(hostDataPtr, xSize * sizeof(float), xSize, ySize);
			hipExtent extent = make_hipExtent(xSize * sizeof(float), ySize, zSize);
			hipMemcpy3DParms memcopyParameters = { 0 };
			memcopyParameters.srcPtr = devicePtr;
			memcopyParameters.dstPtr = hostPtr;
			memcopyParameters.extent = extent;
			memcopyParameters.kind = hipMemcpyDeviceToHost;
			hipMemcpy3D(&memcopyParameters);
			return std::shared_ptr<float>(hostDataPtr, std::default_delete<float[]>());
		}

		__device__ void setVolumeElement(hipPitchedPtr volumePtr, size_t ySize, size_t xCoord, size_t yCoord, size_t zCoord, float value) {
			char* devicePtr = (char*)(volumePtr.ptr);
			//z * xSize * ySize + y * xSize + x
			size_t pitch = volumePtr.pitch;
			size_t slicePitch = pitch * ySize;
			char* slice = devicePtr + zCoord*slicePitch;
			float* row = (float*)(slice + yCoord * pitch);
			row[xCoord] = value;
		}

		__global__ void reconstructionKernel(cv::cuda::PtrStepSz<float> const& image, hipPitchedPtr volumePtr, size_t xSize, size_t ySize, size_t zSize, double radiusSquared, double sine, double cosine, double heightOffset, double uOffset, double SD, double imageLowerBoundU, double imageUpperBoundU, double imageLowerBoundV, double imageUpperBountV, double volumeToWorldXPrecomputed, double volumeToWorldYPrecomputed, double volumeToWorldZPrecomputed, double imageToMatUPrecomputed, double imageToMatVPrecomputed) {
			size_t x = blockIdx.x;
			size_t y = blockIdx.y;
			size_t z = blockIdx.z;
			if (x < xSize && y < ySize && z < zSize) {
				float value = std::sqrt(double(x*x + y*y + z*z));
				setVolumeElement(volumePtr, ySize, x, y, z, value);
			}
		}

		void startReconstruction(cv::cuda::PtrStepSz<float> const & image, hipPitchedPtr volumePtr, size_t xSize, size_t ySize, size_t zSize, double radiusSquared, double sine, double cosine, double heightOffset, double uOffset, double SD, double imageLowerBoundU, double imageUpperBoundU, double imageLowerBoundV, double imageUpperBoundV, double volumeToWorldXPrecomputed, double volumeToWorldYPrecomputed, double volumeToWorldZPrecomputed, double imageToMatUPrecomputed, double imageToMatVPrecomputed) {
			dim3 blocks(xSize, ySize, zSize);
			reconstructionKernel <<< blocks, 1 >>>(image,
												   volumePtr,
												   xSize,
												   ySize,
												   zSize,
												   radiusSquared,
												   sine,
												   cosine,
												   heightOffset,
												   uOffset,
												   SD,
												   imageLowerBoundU,
												   imageUpperBoundU,
												   imageLowerBoundV,
												   imageUpperBoundV,
												   volumeToWorldXPrecomputed,
												   volumeToWorldYPrecomputed,
												   volumeToWorldZPrecomputed,
												   imageToMatUPrecomputed,
												   imageToMatVPrecomputed);
		}
		
	}

}