#include "hip/hip_runtime.h"
#include "cuda_ct.h"
#include "hip/hip_math_constants.h"

namespace ct {

	namespace cuda {

		size_t getFreeMemory() {
			size_t freeMemory, totalMemory;
			hipMemGetInfo(&freeMemory, &totalMemory);
			return freeMemory;
		}

		size_t getTotalMemory() {
			size_t freeMemory, totalMemory;
			hipMemGetInfo(&freeMemory, &totalMemory);
			return totalMemory;
		}

		int getMultiprocessorCnt(int deviceId) {
			int cnt;
			hipDeviceGetAttribute(&cnt, hipDeviceAttributeMultiprocessorCount, deviceId);
			return cnt;
		}

		int getMemoryBusWidth(int deviceId) {
			int busWidth;
			hipDeviceGetAttribute(&busWidth, hipDeviceAttributeMemoryBusWidth, deviceId);
			return busWidth;
		}

		int getMemoryClockRate(int deviceId) {
			int busWidth;
			hipDeviceGetAttribute(&busWidth, hipDeviceAttributeMemoryBusWidth, deviceId);
			return busWidth;
		}

		std::string getDeviceName(int deviceId) {
			hipSetDevice(deviceId);
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, deviceId);
			double memory = getTotalMemory();
			std::ostringstream out;
			out << std::setprecision(1) << std::fixed << prop.name << ", " << memory / 1024.0 / 1024.0 / 1024.0 << " Gb";
			return out.str();
		}

		__device__ float ramLakWindowFilter(float n, float N){
			return n / N;
		}

		__device__ float sheppLoganWindowFilter(float n, float N) {
			if (n == 0) {
				return 0.0f;
			} else {
				float rl = ramLakWindowFilter(n, N);
				return (rl)* (__sinf(rl*0.5f*HIP_PI_F)) / (rl*0.5f*HIP_PI_F);
			}

		}

		__device__ float hannWindowFilter(float n, float N) {
			return ramLakWindowFilter(n, N) * 0.5f*(1.0f + __cosf((2.0f * HIP_PI_F * float(n)) / (float(N) * 2.0f)));
		}

		__global__ void frequencyFilterKernel(cv::cuda::PtrStepSz<float2> image, int filterType) {

			size_t xIndex = threadIdx.x + blockIdx.x * blockDim.x;
			size_t yIndex = threadIdx.y + blockIdx.y * blockDim.y;

			if (xIndex < image.cols && yIndex < image.rows) {
				float2 pixel = image(yIndex, xIndex);
				float factor;
				if (filterType == 0) {
					factor = ramLakWindowFilter(xIndex, image.cols);
				} else if (filterType == 1) {
					factor = sheppLoganWindowFilter(xIndex, image.cols);
				} else if (filterType == 2) {
					factor = hannWindowFilter(xIndex, image.cols);
				}
				image(yIndex, xIndex) = make_float2(pixel.x*factor, pixel.y*factor);
			}

		}

		void applyFrequencyFiltering(cv::cuda::PtrStepSz<float2> image, int filterType, hipStream_t stream, bool& success) {
			success = true;
			dim3 threads(32, 1);
			dim3 blocks(std::ceil(float(image.cols) / float(threads.x)),
						std::ceil(float(image.rows) / float(threads.y)));
			frequencyFilterKernel << < blocks, threads, 0, stream >> >(image, filterType);

			hipError_t status = hipGetLastError();
			if (status != hipSuccess) {
				std::cout << std::endl << "Kernel launch ERROR: " << hipGetErrorString(status);
				success = false;
			}
		}

		__device__ float W(float D, float u, float v) {
			return D * rsqrtf(D*D + u*u + v*v);
		}

		__global__ void feldkampWeightFilterKernel(cv::cuda::PtrStepSz<float> image, float SD, float matToImageUPreprocessed, float matToImageVPreprocessed) {

			size_t xIndex = threadIdx.x + blockIdx.x * blockDim.x;
			size_t yIndex = threadIdx.y + blockIdx.y * blockDim.y;

			if (xIndex < image.cols && yIndex < image.rows) {
				float u = float(xIndex) - matToImageUPreprocessed;
				float v = -float(yIndex) + matToImageVPreprocessed;
				image(yIndex, xIndex) *= W(SD, u, v);
			}

		}

		void applyFeldkampWeightFiltering(cv::cuda::PtrStepSz<float> image, float SD, float matToImageUPreprocessed, float matToImageVPreprocessed, hipStream_t stream, bool& success) {
			success = true;
			dim3 threads(32, 1);
			dim3 blocks(std::ceil(float(image.cols) / float(threads.x)),
						std::ceil(float(image.rows) / float(threads.y)));
			feldkampWeightFilterKernel << < blocks, threads, 0, stream >> >(image, SD, matToImageUPreprocessed, matToImageVPreprocessed);

			hipError_t status = hipGetLastError();
			if (status != hipSuccess) {
				std::cout << std::endl << "Kernel launch ERROR: " << hipGetErrorString(status);
				success = false;
			}
		}

		hipPitchedPtr create3dVolumeOnGPU(size_t xSize, size_t ySize, size_t zSize, bool& success) {
			success = true;
			hipError_t status;
			hipExtent extent = make_hipExtent(xSize * sizeof(float), ySize, zSize);
			hipPitchedPtr ptr;
			status = hipMalloc3D(&ptr, extent);
			if (status != hipSuccess) {
				std::cout << "hipMalloc3D ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
			status = hipMemset3D(ptr, 0, extent);
			if (status != hipSuccess) {
				std::cout << "hipMemset3D ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
			//if something went wrong try to deallocate memory
			if (!success) hipFree(ptr.ptr);
			return ptr;
		}

		void delete3dVolumeOnGPU(hipPitchedPtr devicePtr, bool& success) {
			success = true;
			hipError_t status = hipFree(devicePtr.ptr);
			if (status != hipSuccess) {
				std::cout << "hipFree ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
		}

		void download3dVolume(hipPitchedPtr devicePtr, float* hostPtr, size_t xSize, size_t ySize, size_t zSize,  bool& success) {
			success = true;
			hipPitchedPtr hostPitchedPtr = make_hipPitchedPtr(hostPtr, xSize * sizeof(float), xSize, ySize);
			hipExtent extent = make_hipExtent(xSize * sizeof(float), ySize, zSize);
			hipMemcpy3DParms memcopyParameters = { 0 };
			memcopyParameters.srcPtr = devicePtr;
			memcopyParameters.dstPtr = hostPitchedPtr;
			memcopyParameters.extent = extent;
			memcopyParameters.kind = hipMemcpyDeviceToHost;
			hipError_t status = hipMemcpy3D(&memcopyParameters);
			if (status != hipSuccess) {
				std::cout << "hipMemcpy3D ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
		}

		__device__ float bilinearInterpolation(float u, float v, float u0v0, float u1v0, float u0v1, float u1v1) {
			//the two interpolations on the u axis
			float v0 = (1.0f - u)*u0v0 + u*u1v0;
			float v1 = (1.0f - u)*u0v1 + u*u1v1;
			//interpolation on the v axis between the two u-interpolated values
			return (1.0f - v)*v0 + v*v1;
		}

		__device__ void addToVolumeElement(hipPitchedPtr volumePtr, size_t xCoord, size_t yCoord, size_t zCoord, float value) {
			char* devicePtr = (char*)(volumePtr.ptr);
			//z * xSize * ySize + y * xSize + x
			size_t pitch = volumePtr.pitch;
			size_t slicePitch = pitch * volumePtr.ysize;
			char* slice = devicePtr + zCoord*slicePitch;
			float* row = (float*)(slice + yCoord * pitch);
			row[xCoord] += value;
		}

		__global__ void reconstructionKernel(cv::cuda::PtrStepSz<float> image, 
											 hipPitchedPtr volumePtr, 
											 size_t xSize, size_t ySize, 
											 size_t zSize, size_t zOffset, 
											 float radiusSquared,
											 float sine,
											 float cosine,
											 float heightOffset,
											 float uOffset,
											 float SD,
											 float imageLowerBoundU,
											 float imageUpperBoundU,
											 float imageLowerBoundV,
											 float imageUpperBoundV,
											 float volumeToWorldXPrecomputed,
											 float volumeToWorldYPrecomputed,
											 float volumeToWorldZPrecomputed,
											 float imageToMatUPrecomputed,
											 float imageToMatVPrecomputed) {

			size_t xIndex = threadIdx.x + blockIdx.x * blockDim.x;
			size_t yIndex = threadIdx.y + blockIdx.y * blockDim.y;
			size_t zIndex = threadIdx.z + blockIdx.z * blockDim.z;

			//if (xIndex == 0 && yIndex == 0 && zIndex == 0) {
			//	printf("kernel start\n");
			//}

			//just make sure we're inside the volume bounds
			if (xIndex < xSize && yIndex < ySize && zIndex < zSize) {

				//calculate the world coordinates
				float x = float(xIndex) - volumeToWorldXPrecomputed;
				float y = float(yIndex) - volumeToWorldYPrecomputed;
				float z = float(zIndex + zOffset) - volumeToWorldZPrecomputed;

				//check if voxel is inside the reconstructable cylinder
				if ((x*x + y*y) < radiusSquared) {

					float t = -x*sine + y*cosine;
					t += uOffset;
					float s = x*cosine + y*sine;
					float u = (t*SD) / (SD - s);
					float v = ((z + heightOffset)*SD) / (SD - s);

					//check if it's inside the image (before the coordinate transformation)
					if (u >= imageLowerBoundU && u <= imageUpperBoundU && v >= imageLowerBoundV && v <= imageUpperBoundV) {

						u += imageToMatUPrecomputed;
						v = -v + imageToMatVPrecomputed;

						//get the 4 surrounding pixels for bilinear interpolation (note: u and v are always positive)
						size_t u0 = u;
						size_t u1 = u0 + 1;
						size_t v0 = v;
						size_t v1 = v0 + 1;

						float u0v0 = image(v0, u0);
						float u1v0 = image(v0, u1);
						float u0v1 = image(v1, u0);
						float u1v1 = image(v1, u1);

						float value = bilinearInterpolation(u - float(u0), v - float(v0), u0v0, u1v0, u0v1, u1v1);

						addToVolumeElement(volumePtr, xIndex, yIndex, zIndex, value);
					}
				}
			}

			//if (xIndex == 0 && yIndex == 0 && zIndex == 0) {
			//	printf("kernel end\n");
			//}

		}

		void startReconstruction(cv::cuda::PtrStepSz<float> image, 
								 hipPitchedPtr volumePtr, 
								 size_t xSize, 
								 size_t ySize, 
								 size_t zSize, 
								 size_t zOffset, 
								 float radiusSquared, 
								 float sine,
								 float cosine,
								 float heightOffset,
								 float uOffset,
								 float SD,
								 float imageLowerBoundU,
								 float imageUpperBoundU,
								 float imageLowerBoundV,
								 float imageUpperBoundV,
								 float volumeToWorldXPrecomputed,
								 float volumeToWorldYPrecomputed,
								 float volumeToWorldZPrecomputed,
								 float imageToMatUPrecomputed,
								 float imageToMatVPrecomputed,
								 hipStream_t stream,
								 bool& success) {
			success = true;
			dim3 threads(16, 16, 1);
			dim3 blocks(std::ceil(float(xSize) / float(threads.x)),
						std::ceil(float(ySize) / float(threads.y)),
						std::ceil(float(zSize) / float(threads.z)));
			reconstructionKernel << < blocks, threads, 0, stream >> >(image,
																	  volumePtr,
																	  xSize,
																	  ySize,
																	  zSize,
																	  zOffset,
																	  radiusSquared,
																	  sine,
																	  cosine,
																	  heightOffset,
																	  uOffset,
																	  SD,
																	  imageLowerBoundU,
																	  imageUpperBoundU,
																	  imageLowerBoundV,
																	  imageUpperBoundV,
																	  volumeToWorldXPrecomputed,
																	  volumeToWorldYPrecomputed,
																	  volumeToWorldZPrecomputed,
																	  imageToMatUPrecomputed,
																	  imageToMatVPrecomputed);
			hipError_t status = hipGetLastError();
			if (status != hipSuccess) {
				std::cout << std::endl << "Kernel launch ERROR: " << hipGetErrorString(status);
				success = false;
			}
		}

	}

}