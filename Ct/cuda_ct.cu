#include "hip/hip_runtime.h"
#include "cuda_ct.h"
#include "hip/hip_math_constants.h"

namespace ct {

	namespace cuda {

		size_t getFreeMemory() {
			size_t freeMemory, totalMemory;
			hipMemGetInfo(&freeMemory, &totalMemory);
			return freeMemory;
		}

		size_t getTotalMemory() {
			size_t freeMemory, totalMemory;
			hipMemGetInfo(&freeMemory, &totalMemory);
			return totalMemory;
		}

		int getMultiprocessorCnt(int deviceId) {
			int cnt;
			hipDeviceGetAttribute(&cnt, hipDeviceAttributeMultiprocessorCount, deviceId);
			return cnt;
		}

		int getMemoryBusWidth(int deviceId) {
			int busWidth;
			hipDeviceGetAttribute(&busWidth, hipDeviceAttributeMemoryBusWidth, deviceId);
			return busWidth;
		}

		int getMemoryClockRate(int deviceId) {
			int clockRate;
			hipDeviceGetAttribute(&clockRate, hipDeviceAttributeMemoryClockRate, deviceId);
			return clockRate;
		}

		std::string getDeviceName(int deviceId) {
			hipSetDevice(deviceId);
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, deviceId);
			double memory = getTotalMemory();
			std::ostringstream out;
			out << std::setprecision(1) << std::fixed << prop.name << ", " << memory / 1024.0 / 1024.0 / 1024.0 << " Gb";
			return out.str();
		}

		__device__ float ramLakWindowFilter(float n, float Nreciprocal){
			return n * Nreciprocal;
		}

		__device__ float sheppLoganWindowFilter(float n, float Nreciprocal) {
			if (n == 0.0f) {
				return 0.0f;
			} else {
				float rl = ramLakWindowFilter(n, Nreciprocal);
				return (rl)* (__sinf(rl*0.5f*HIP_PI_F)) / (rl*0.5f*HIP_PI_F);
			}

		}

		__device__ float hannWindowFilter(float n, float Nreciprocal) {
			float rl = ramLakWindowFilter(n, Nreciprocal);
			return rl * 0.5f*(1.0f + __cosf(HIP_PI_F * rl));
		}

		__global__ void frequencyFilterKernel(cv::cuda::PtrStepSz<float2> image, int filterType) {

			size_t xIndex = threadIdx.x + blockIdx.x * blockDim.x;
			size_t yIndex = threadIdx.y + blockIdx.y * blockDim.y;

			if (xIndex < image.cols && yIndex < image.rows) {
				float2 pixel = image(yIndex, xIndex);
				float Nreciprocal = 1.0 / static_cast<float>(image.cols);
				float factor;
				if (filterType == 0) {
					factor = ramLakWindowFilter(xIndex, Nreciprocal);
				} else if (filterType == 1) {
					factor = sheppLoganWindowFilter(xIndex, Nreciprocal);
				} else if (filterType == 2) {
					factor = hannWindowFilter(xIndex, Nreciprocal);
				}
				image(yIndex, xIndex) = make_float2(pixel.x*factor, pixel.y*factor);
			}

		}

		void applyFrequencyFiltering(cv::cuda::PtrStepSz<float2> image, int filterType, hipStream_t stream, bool& success) {
			success = true;
			dim3 threads(32, 1);
			dim3 blocks(std::ceil(float(image.cols) / float(threads.x)),
						std::ceil(float(image.rows) / float(threads.y)));
			frequencyFilterKernel << < blocks, threads, 0, stream >> >(image, filterType);

			hipError_t status = hipGetLastError();
			if (status != hipSuccess) {
				std::cout << std::endl << "Kernel launch ERROR: " << hipGetErrorString(status);
				success = false;
			}
		}

		__device__ float W(float D, float u, float v) {
			return D * rsqrtf(D*D + u*u + v*v);
		}

		__global__ void feldkampWeightFilterKernel(cv::cuda::PtrStepSz<float> image, float SD, float uPrecomputed, float vPrecomputed) {

			size_t xIndex = threadIdx.x + blockIdx.x * blockDim.x;
			size_t yIndex = threadIdx.y + blockIdx.y * blockDim.y;

			if (xIndex < image.cols && yIndex < image.rows) {
				float u = float(xIndex) - uPrecomputed;
				float v = -float(yIndex) + vPrecomputed;
				image(yIndex, xIndex) *= W(SD, u, v);
			}

		}

		void applyFeldkampWeightFiltering(cv::cuda::PtrStepSz<float> image, float SD, float uPrecomputed, float vPrecomputed, hipStream_t stream, bool& success) {
			success = true;
			dim3 threads(32, 1);
			dim3 blocks(std::ceil(float(image.cols) / float(threads.x)),
						std::ceil(float(image.rows) / float(threads.y)));
			feldkampWeightFilterKernel << < blocks, threads, 0, stream >> >(image, SD, uPrecomputed, vPrecomputed);

			hipError_t status = hipGetLastError();
			if (status != hipSuccess) {
				std::cout << std::endl << "Kernel launch ERROR: " << hipGetErrorString(status);
				success = false;
			}
		}

		hipPitchedPtr create3dVolumeOnGPU(size_t xSize, size_t ySize, size_t zSize, bool& success, bool verbose) {
			success = true;
			hipError_t status;
			hipExtent extent = make_hipExtent(xSize * sizeof(float), ySize, zSize);
			hipPitchedPtr ptr;
			status = hipMalloc3D(&ptr, extent);
			if (status != hipSuccess) {
				if (verbose) std::cout << "hipMalloc3D ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
			//if something went wrong try to deallocate memory
			if (!success) hipFree(ptr.ptr);
			return ptr;
		}

		void setToZero(hipPitchedPtr devicePtr, size_t xSize, size_t ySize, size_t zSize, bool& success) {
			success = true;
			hipError_t status;
			hipExtent extent = make_hipExtent(xSize * sizeof(float), ySize, zSize);		
			status = hipMemset3D(devicePtr, 0, extent);
			if (status != hipSuccess) {
				std::cout << "hipMemset3D ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
		}

		void delete3dVolumeOnGPU(hipPitchedPtr devicePtr, bool& success) {
			success = true;
			hipError_t status = hipFree(devicePtr.ptr);
			if (status != hipSuccess) {
				std::cout << "hipFree ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
		}

		void download3dVolume(hipPitchedPtr devicePtr, float* hostPtr, size_t xSize, size_t ySize, size_t zSize,  bool& success) {
			success = true;
			hipPitchedPtr hostPitchedPtr = make_hipPitchedPtr(hostPtr, xSize * sizeof(float), xSize, ySize);
			hipExtent extent = make_hipExtent(xSize * sizeof(float), ySize, zSize);
			hipMemcpy3DParms memcopyParameters = { 0 };
			memcopyParameters.srcPtr = devicePtr;
			memcopyParameters.dstPtr = hostPitchedPtr;
			memcopyParameters.extent = extent;
			memcopyParameters.kind = hipMemcpyDeviceToHost;
			hipError_t status = hipMemcpy3D(&memcopyParameters);
			if (status != hipSuccess) {
				std::cout << "hipMemcpy3D ERROR: " << hipGetErrorString(status) << std::endl;
				success = false;
			}
		}

		__device__ float bilinearInterpolation(float u, float v, float u0v0, float u1v0, float u0v1, float u1v1) {
			//the two interpolations on the u axis
			float v0 = (1.0f - u)*u0v0 + u*u1v0;
			float v1 = (1.0f - u)*u0v1 + u*u1v1;
			//interpolation on the v axis between the two u-interpolated values
			return (1.0f - v)*v0 + v*v1;
		}

		__device__ void addToVolumeElement(hipPitchedPtr volumePtr, size_t xCoord, size_t yCoord, size_t zCoord, float value) {
			char* devicePtr = (char*)(volumePtr.ptr);
			//z * xSize * ySize + y * xSize + x
			size_t pitch = volumePtr.pitch;
			size_t slicePitch = pitch * volumePtr.ysize;
			char* slice = devicePtr + zCoord*slicePitch;
			float* row = (float*)(slice + yCoord * pitch);
			row[xCoord] += value;
		}

		__global__ void reconstructionKernel(cv::cuda::PtrStepSz<float> image, 
											 hipPitchedPtr volumePtr, 
											 size_t xSize, size_t ySize, 
											 size_t zSize, size_t zOffset, 
											 float radiusSquared,
											 float sine,
											 float cosine,
											 float heightOffset,
											 float uOffset,
											 float SD,
											 float imageLowerBoundU,
											 float imageUpperBoundU,
											 float imageLowerBoundV,
											 float imageUpperBoundV,
											 float xPrecomputed,
											 float yPrecomputed,
											 float zPrecomputed,
											 float uPrecomputed,
											 float vPrecomputed) {

			size_t xIndex = threadIdx.x + blockIdx.x * blockDim.x;
			size_t yIndex = threadIdx.y + blockIdx.y * blockDim.y;
			size_t zIndex = threadIdx.z + blockIdx.z * blockDim.z;

			//if (xIndex == 0 && yIndex == 0 && zIndex == 0) {
			//	printf("kernel start\n");
			//}

			//just make sure we're inside the volume bounds
			if (xIndex < xSize && yIndex < ySize && zIndex < zSize) {

				//calculate the world coordinates
				float x = float(xIndex) - xPrecomputed;
				float y = float(yIndex) - yPrecomputed;
				float z = float(zIndex + zOffset) - zPrecomputed;

				//check if voxel is inside the reconstructable cylinder
				if ((x*x + y*y) < radiusSquared) {

					float t = -x*sine + y*cosine;
					t += uOffset;
					float s = x*cosine + y*sine;
					float u = (t*SD) / (SD - s);
					float v = ((z + heightOffset)*SD) / (SD - s);

					//check if it's inside the image (before the coordinate transformation)
					if (u >= imageLowerBoundU && u <= imageUpperBoundU && v >= imageLowerBoundV && v <= imageUpperBoundV) {

						//calculate weight
						float w = SD / (SD + s);
						w = w*w;

						u += uPrecomputed;
						v = -v + vPrecomputed;

						//get the 4 surrounding pixels for bilinear interpolation (note: u and v are always positive)
						size_t u0 = u;
						size_t u1 = u0 + 1;
						size_t v0 = v;
						size_t v1 = v0 + 1;

						float* row = image.ptr(v0);
						float u0v0 = row[u0];
						float u1v0 = row[u1];
						row = image.ptr(v1);
						float u0v1 = row[u0];
						float u1v1 = row[u1];

						float value = w * bilinearInterpolation(u - float(u0), v - float(v0), u0v0, u1v0, u0v1, u1v1);

						addToVolumeElement(volumePtr, xIndex, yIndex, zIndex, value);
					}
				}
			}

			//if (xIndex == 0 && yIndex == 0 && zIndex == 0) {
			//	printf("kernel end\n");
			//}

		}

		void startReconstruction(cv::cuda::PtrStepSz<float> image, 
								 hipPitchedPtr volumePtr, 
								 size_t xSize, 
								 size_t ySize, 
								 size_t zSize, 
								 size_t zOffset, 
								 float radiusSquared, 
								 float sine,
								 float cosine,
								 float heightOffset,
								 float uOffset,
								 float SD,
								 float imageLowerBoundU,
								 float imageUpperBoundU,
								 float imageLowerBoundV,
								 float imageUpperBoundV,
								 float xPrecomputed,
								 float yPrecomputed,
								 float zPrecomputed,
								 float uPrecomputed,
								 float vPrecomputed,
								 hipStream_t stream,
								 bool& success) {
			success = true;
			dim3 threads(16, 16, 1);
			dim3 blocks(std::ceil(float(xSize) / float(threads.x)),
						std::ceil(float(ySize) / float(threads.y)),
						std::ceil(float(zSize) / float(threads.z)));
			reconstructionKernel << < blocks, threads, 0, stream >> >(image,
																	  volumePtr,
																	  xSize,
																	  ySize,
																	  zSize,
																	  zOffset,
																	  radiusSquared,
																	  sine,
																	  cosine,
																	  heightOffset,
																	  uOffset,
																	  SD,
																	  imageLowerBoundU,
																	  imageUpperBoundU,
																	  imageLowerBoundV,
																	  imageUpperBoundV,
																	  xPrecomputed,
																	  yPrecomputed,
																	  zPrecomputed,
																	  uPrecomputed,
																	  vPrecomputed);
			hipError_t status = hipGetLastError();
			if (status != hipSuccess) {
				std::cout << std::endl << "Kernel launch ERROR: " << hipGetErrorString(status);
				success = false;
			}
		}

	}

}